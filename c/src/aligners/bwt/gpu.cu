
extern "C" {
#include "gpu.h"
}

#include "BW_gpu.cuh"

//-----------------------------------------------------------------------------
// GPU context functions
//-----------------------------------------------------------------------------

gpu_context_t *gpu_context_new(unsigned int id, unsigned int num_threads,
			       bwt_index_t *index) {
  gpu_context_t *context = (gpu_context_t*) calloc(1, sizeof(gpu_context_t));

  // initializes
  context->id = id;
  context->num_threads = num_threads;

  context->d_nWe = NULL; 
  context->d_nWe_size = 0;

  context->d_We = NULL;
  context->d_We_size = 0;

  context->d_k = NULL;
  context->d_k_size = 0;

  context->d_l = NULL;
  context->d_l_size = 0;

  // hipSetDevice
  gpu_set_device(id);

  // copyVectorGPU
  gpu_copy_vector(&context->d_C,   &index->h_C,   sizeof(size_t));
  gpu_copy_vector(&context->d_C1,  &index->h_C1,  sizeof(size_t));
  gpu_copy_vector(&context->d_rC,  &index->h_rC,  sizeof(size_t));
  gpu_copy_vector(&context->d_rC1, &index->h_rC1, sizeof(size_t));

  // copyCompMatrixGPU
  gpu_copy_matrix(&context->d_O, &index->h_O);
  reverseStrandO(&context->d_rO, &context->d_O);

  return context;
}

//-----------------------------------------------------------------------------

void gpu_context_free(gpu_context_t *context) {
  if (context != NULL) return;

  // free memory                                                  
  if (context->d_We != NULL) {
    hipFree(context->d_We);
    //manageCudaError();
  }


  if (context->d_nWe != NULL) {
    hipFree(context->d_nWe);
    //manageCudaError();
  }

  if (context->d_k != NULL) {
    hipFree(context->d_k);
    //manageCudaError();
  }

  if (context->d_l != NULL) {
    hipFree(context->d_l);
    //manageCudaError();
  }

  free(context);
}

//-----------------------------------------------------------------------------
// set device
//-----------------------------------------------------------------------------

extern "C" void gpu_set_device(unsigned int id) {
     hipSetDevice(id);
}

//-----------------------------------------------------------------------------
// copỳ vector and matrix to device
//-----------------------------------------------------------------------------

extern "C" void gpu_copy_vector(vector *dest, vector *src, size_t size) {
  copyVectorGPU(dest, src, size);
}

extern "C" void gpu_copy_matrix(comp_matrix *dest, comp_matrix *src) {
  copyCompMatrixGPU(dest, src);
}

//-----------------------------------------------------------------------------
// reallocate memory in device
//-----------------------------------------------------------------------------

extern "C" void gpu_reallocate_memory(size_t *new_size, size_t *old_size, void **data) {
  if (*old_size < *new_size) {
    if (*data != NULL) {
      hipFree(*data);
      //manageCudaError();
    }
    hipMalloc((void**) data, *new_size);
    //manageCudaError();
    *old_size = *new_size;
  } 
}

//-----------------------------------------------------------------------------
// gpu get k and l values
//-----------------------------------------------------------------------------

extern "C" void gpu_get_kl_values(size_t seed_size, size_t min_seed_size, 
       	   			  size_t num_max_seeds,
				  size_t num_reads, size_t seqs_size,
				  char *seqs, size_t *indices,
				  gpu_context_t *context, 
				  size_t *k_values, size_t *l_values,
				  unsigned char mode) {
  
  unsigned int num_threads = context->num_threads;
  unsigned int num_blocks = (num_reads / num_threads) + ((num_reads % num_threads == 0) ? 0 : 1);
  
  // allocate memory in device (GPU) for inputs
  gpu_reallocate_memory(&seqs_size, &context->d_We_size, (void **) &context->d_We);

  size_t indices_size = (num_reads + 1) * sizeof(size_t);
  gpu_reallocate_memory(&indices_size, &context->d_nWe_size, (void **) &context->d_nWe);
  size_t kl_size;

  if (mode == SEED_MODE) {
    kl_size = num_reads * (2 * num_max_seeds) * sizeof(size_t);
  }else {  
    kl_size = num_reads * sizeof(size_t);
  }

  //printf("KL %i\n", num_reads * (2 * num_max_seeds));

  gpu_reallocate_memory(&kl_size, &context->d_k_size, (void **) &context->d_k);
  gpu_reallocate_memory(&kl_size, &context->d_l_size, (void **) &context->d_l);
  
  // copy from host to device (CPU -> GPU)
  hipMemcpy(context->d_We, seqs, seqs_size, hipMemcpyHostToDevice);
  //manageCudaError();
  hipMemcpy(context->d_nWe, indices, indices_size, hipMemcpyHostToDevice);
  //manageCudaError();

  // call CUDA kernels and copy gpu results to cpu and insert them to list to be processed    

  // searching with d_O (normal strand)
  if (mode == SEED_MODE) {
     //printf("SEED MODE :\n Num Reads: %i\n Seed Size: %i\n Min Seed Size: %i\n Num Max Seeds: %i\n", num_reads, seed_size, min_seed_size, num_max_seeds);

     BWExactSearchBackwardGPUSeedsWrapper_ex(num_blocks, num_threads, context->d_We, context->d_nWe, 
     					context->d_k, context->d_l, 0, context->d_O.siz-2, 
				        &context->d_C, &context->d_C1, &context->d_O, num_reads, 
					seed_size, min_seed_size, num_max_seeds);
  }else {
     BWExactSearchBackwardGPUWrapper_ex(num_blocks, num_threads, context->d_We, context->d_nWe, 
     					context->d_k, context->d_l, 0, context->d_O.siz-2, 
				        &context->d_C, &context->d_C1, &context->d_O, num_reads);
  }

  // copy back the results (GPU -> CPU);
  hipMemcpy(k_values, context->d_k, kl_size, hipMemcpyDeviceToHost);
  //manageCudaError();
  hipMemcpy(l_values, context->d_l, kl_size, hipMemcpyDeviceToHost);
  //manageCudaError();


  // searching with d_rO (reverse strand)
  if (mode == SEED_MODE) {
     //printf("SEED MODE\n");
     BWExactSearchForwardGPUSeedsWrapper_ex(num_blocks, num_threads, context->d_We, 
					    context->d_nWe,	
					    context->d_k, context->d_l, 0, 
					    context->d_O.siz-2, &context->d_rC, 
					    &context->d_rC1, &context->d_rO, num_reads, 
					    seed_size, min_seed_size, num_max_seeds);
     // copy back the results (GPU -> CPU);
     hipMemcpy(k_values + (num_reads * (2 * num_max_seeds)), context->d_k, kl_size, hipMemcpyDeviceToHost);
     //manageCudaError();
     hipMemcpy(l_values + (num_reads * (2 * num_max_seeds)), context->d_l, kl_size, hipMemcpyDeviceToHost);
     //manageCudaError();
     
  }else {
     BWExactSearchForwardGPUWrapper_ex(num_blocks, num_threads, context->d_We, context->d_nWe,	
     				       context->d_k, context->d_l, 0, context->d_O.siz-2, 
                                       &context->d_rC, &context->d_rC1, &context->d_rO, num_reads);
     // copy back the results (GPU -> CPU);
     hipMemcpy(k_values + num_reads, context->d_k, kl_size, hipMemcpyDeviceToHost);
     //manageCudaError();
     hipMemcpy(l_values + num_reads, context->d_l, kl_size, hipMemcpyDeviceToHost);
     //manageCudaError();
     
  }

}


//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
